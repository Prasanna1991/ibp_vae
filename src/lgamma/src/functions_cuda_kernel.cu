#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include "functions_cuda_kernel.h"
#include "internals.h"
#include <math.h>
#include <unistd.h>
#include <stdlib.h>

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define real float
#define NUM_BLOCKS 256

__device__ hiprandState_t global_states[256];

// there's a way to write shorter code by templating float/double, but without knowing much about template overhead (which I think is small, but not certain) I'm just going to reimplement + vim

__global__ void polygamma_cuda_kernel(int n, int input_sheight, int input_swidth, int output_sheight, int output_swidth, int height, int width, float *input_data, float *output_data) {
  for (int addr = threadIdx.x; addr < width * height; addr += blockDim.x)
    output_data[addr] = polygamma_impl(n, input_data[addr]);
}

__global__ void lgamma_cuda_kernel(int input_sheight, int input_swidth, int output_sheight, int output_swidth, int height, int width, float *input_data, float *output_data) {
  for (int addr = threadIdx.x; addr < width * height; addr += blockDim.x)
    output_data[addr] = lgamma(input_data[addr]);
}

__global__ void lbeta_cuda_kernel(int a_sheight, int a_swidth, int b_sheight, int b_swidth, int output_sheight, int output_swidth, int height, int width, float *a_data, float *b_data, float *output_data) {
  for (int addr = threadIdx.x; addr < width * height; addr += blockDim.x)
    output_data[addr] = lbeta_impl(a_data[addr], b_data[addr]);
}

__global__ void polygamma_cuda_dbl_kernel(int n, int input_sheight, int input_swidth, int output_sheight, int output_swidth, int height, int width, double *input_data, double *output_data) {
  for (int addr = threadIdx.x; addr < width * height; addr += blockDim.x)
    output_data[addr] = polygamma_impl_dbl(n, input_data[addr]);
}

__global__ void lgamma_cuda_dbl_kernel(int input_sheight, int input_swidth, int output_sheight, int output_swidth, int height, int width, double *input_data, double *output_data) {
  for (int addr = threadIdx.x; addr < width * height; addr += blockDim.x)
    output_data[addr] = lgamma(input_data[addr]);
}

__global__ void lbeta_cuda_dbl_kernel(int a_sheight, int a_swidth, int b_sheight, int b_swidth, int output_sheight, int output_swidth, int height, int width, double *a_data, double *b_data, double *output_data) {
  for (int addr = threadIdx.x; addr < width * height; addr += blockDim.x)
    output_data[addr] = lbeta_impl_dbl(a_data[addr], b_data[addr]);
}

__global__ void init(unsigned int seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &global_states[idx]);     
    __syncthreads();
}

// just for compilation purposes - this is Marsaglia's algorithm
__global__ void sample_gamma_dbl_kernel(int height, int width, double *a_data, double *output_data) {
  for (int addr = threadIdx.x; addr < width * height; addr += blockDim.x) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    double d = a_data[addr]  - (1./3.);
    double c = 1./sqrt(9. * d);
    double u, v, x = 0;
    do {
      x = hiprand_normal(&global_states[idx]);
      v = (1 + c * x) * (1 + c * x) * (1 + c * x);
      u = hiprand_uniform(&global_states[idx]);
    } while (v <= 0. || (log(u) >= 0.5 * x * x + d * (1 - v + log(v))));
    output_data[addr] = d * v;
  }
}

#ifdef __cplusplus
extern "C" {
#endif

int polygamma_cuda_wrapped(int n, int input_strideHeight, int input_strideWidth, int output_strideHeight, int output_strideWidth, int height, int width, float *input_data, float *output_data) {
  polygamma_cuda_kernel<<<1, NUM_BLOCKS>>>(n, input_strideHeight, input_strideWidth, output_strideHeight, output_strideWidth, height, width, input_data, output_data);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in polygamma_cuda_kernel: %s\n", hipGetErrorString(err));
    return 1;
  }
  return 0;
}

int lgamma_cuda_wrapped(int input_strideHeight, int input_strideWidth, int output_strideHeight, int output_strideWidth, int height, int width, float *input_data, float *output_data) {
  lgamma_cuda_kernel<<<1, NUM_BLOCKS>>>(input_strideHeight, input_strideWidth, output_strideHeight, output_strideWidth, height, width, input_data, output_data);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in lgamma_cuda_kernel: %s\n", hipGetErrorString(err));
    return 1;
  }
  return 0;
}

int lbeta_cuda_wrapped(int a_strideHeight, int a_strideWidth, int b_strideHeight, int b_strideWidth, int output_strideHeight, int output_strideWidth, int height, int width, float *a_data, float *b_data, float *output_data) {
  lbeta_cuda_kernel<<<1, NUM_BLOCKS>>>(a_strideHeight, a_strideWidth, b_strideHeight, b_strideWidth, output_strideHeight, output_strideWidth, height, width, a_data, b_data, output_data);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in lbeta_cuda_kernel: %s\n", hipGetErrorString(err));
    return 1;
  }
  return 0;
}

int polygamma_cuda_dbl_wrapped(int n, int input_strideHeight, int input_strideWidth, int output_strideHeight, int output_strideWidth, int height, int width, double *input_data, double *output_data) {
  polygamma_cuda_dbl_kernel<<<1, NUM_BLOCKS>>>(n, input_strideHeight, input_strideWidth, output_strideHeight, output_strideWidth, height, width, input_data, output_data);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in polygamma_cuda_dbl_kernel: %s\n", hipGetErrorString(err));
    return 1;
  }
  return 0;
}

int lgamma_cuda_dbl_wrapped(int input_strideHeight, int input_strideWidth, int output_strideHeight, int output_strideWidth, int height, int width, double *input_data, double *output_data) {
  lgamma_cuda_dbl_kernel<<<1, NUM_BLOCKS>>>(input_strideHeight, input_strideWidth, output_strideHeight, output_strideWidth, height, width, input_data, output_data);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in lgamma_cuda_dbl_kernel: %s\n", hipGetErrorString(err));
    return 1;
  }
  return 0;
}

int lbeta_cuda_dbl_wrapped(int a_strideHeight, int a_strideWidth, int b_strideHeight, int b_strideWidth, int output_strideHeight, int output_strideWidth, int height, int width, double *a_data, double *b_data, double *output_data) {
  lbeta_cuda_dbl_kernel<<<1, NUM_BLOCKS>>>(a_strideHeight, a_strideWidth, b_strideHeight, b_strideWidth, output_strideHeight, output_strideWidth, height, width, a_data, b_data, output_data);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in lbeta_cuda_dbl_kernel: %s\n", hipGetErrorString(err));
    return 2;
  }
  return 0;
}

int sample_gamma_dbl_wrapped(int height, int width, double *a_data, double *output_data) {
  sample_gamma_dbl_kernel<<<1, NUM_BLOCKS>>>(height, width, a_data, output_data);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in sample_gamma_dbl_wrapped: %s\n", hipGetErrorString(err));
    return 2;
  }
  return 0;
}

void init_rand(void) {
    init<<<1, NUM_BLOCKS>>>(time(NULL));
}

#ifdef __cplusplus
}
#endif
