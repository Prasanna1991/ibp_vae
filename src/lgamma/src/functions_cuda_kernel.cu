#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include "functions_cuda_kernel.h"
#include "internals.h"
#include <math.h>
#include <unistd.h>
#include <stdlib.h>

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define real float
#define NUM_BLOCKS 256

__device__ hiprandState_t global_states[256];

// there's a way to write shorter code by templating float/double, but without knowing much about template overhead (which I think is small, but not certain) I'm just going to reimplement + vim

__global__ void polygamma_cuda_kernel(int n, int input_sheight, int input_swidth, int output_sheight, int output_swidth, int height, int width, float *input_data, float *output_data) {
  for (int addr = threadIdx.x; addr < width * height; addr += blockDim.x)
    output_data[addr] = polygamma_impl(n, input_data[addr]);
}

__global__ void lgamma_cuda_kernel(int input_sheight, int input_swidth, int output_sheight, int output_swidth, int height, int width, float *input_data, float *output_data) {
  for (int addr = threadIdx.x; addr < width * height; addr += blockDim.x)
    output_data[addr] = lgamma(input_data[addr]);
}

__global__ void lbeta_cuda_kernel(int a_sheight, int a_swidth, int b_sheight, int b_swidth, int output_sheight, int output_swidth, int height, int width, float *a_data, float *b_data, float *output_data) {
  for (int addr = threadIdx.x; addr < width * height; addr += blockDim.x)
    output_data[addr] = lbeta_impl(a_data[addr], b_data[addr]);
}

__global__ void polygamma_cuda_dbl_kernel(int n, int input_sheight, int input_swidth, int output_sheight, int output_swidth, int height, int width, double *input_data, double *output_data) {
  for (int addr = threadIdx.x; addr < width * height; addr += blockDim.x)
    output_data[addr] = polygamma_impl_dbl(n, input_data[addr]);
}

__global__ void lgamma_cuda_dbl_kernel(int input_sheight, int input_swidth, int output_sheight, int output_swidth, int height, int width, double *input_data, double *output_data) {
  for (int addr = threadIdx.x; addr < width * height; addr += blockDim.x)
    output_data[addr] = lgamma(input_data[addr]);
}

__global__ void lbeta_cuda_dbl_kernel(int a_sheight, int a_swidth, int b_sheight, int b_swidth, int output_sheight, int output_swidth, int height, int width, double *a_data, double *b_data, double *output_data) {
  for (int addr = threadIdx.x; addr < width * height; addr += blockDim.x)
    output_data[addr] = lbeta_impl_dbl(a_data[addr], b_data[addr]);
}

__global__ void init(unsigned int seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &global_states[idx]);
    __syncthreads();
}

__device__ double sample_expo_internal(hiprandState_t *state) {
    return -log(1.0 - hiprand_uniform(state));
}

__device__ double sample_gamma_internal(hiprandState_t *state, double shape) {
    if (shape < 1.) {
        double u, v, x, y;
        while (1) {
            u = hiprand_uniform(state);
            v = sample_expo_internal(state);
            if (u <= 1.0 - shape) {
                x = pow(u, 1./shape);
                if (x <= v) {
                    return x;
                }
            }
            else {
                y = -log((1 - u)/shape);
                x = pow(1.0 - shape + shape*y, 1./shape);
                if (x <= (v + y)) {
                    return x;
                }
            }
        }
    }
    else if (shape > 1.) {
        double d = shape - (1./3.);
        double c = 1./sqrt(9. * d);
        double u, v, x = 0;
        do {
          x = hiprand_normal(state);
          v = (1 + c * x) * (1 + c * x) * (1 + c * x);
          u = hiprand_uniform(state);
        } while (v <= 0. || ((log(u) >= 0.5 * x * x + d * (1 - v + log(v)))) && (u < 1.0 - 0.0331*(x*x)*(x*x)));
        return d * v;
    }
    else {
        return sample_expo_internal(state);
    }
}

// just for compilation purposes - this is Marsaglia's algorithm
__global__ void sample_gamma_dbl_kernel(int height, int width, double *a_data, double *output_data) {
  for (int addr = threadIdx.x; addr < width * height; addr += blockDim.x) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    output_data[addr] = sample_gamma_internal(&global_states[idx], a_data[addr]);
  }
}

// implementation of Johnk's algorithm
__global__ void sample_beta_dbl_kernel(int height, int width, double *a_data, double *b_data, double *output_data) {
  for (int addr = threadIdx.x; addr < width * height; addr += blockDim.x) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    double a = a_data[addr];
    double b = b_data[addr];
    if ((a <= 1.) && (b <= 1.)) {
        double U, V, X, Y;

        // some spacing issues here, fix TODO
        while (1) {
            U = hiprand_uniform(&global_states[idx]);
            V = hiprand_uniform(&global_states[idx]);
            X = pow(U, 1.0/a);
            Y = pow(V, 1.0/b);

            if ((X + Y) <=  1.0) {
                if (X +Y > 0) {
                    output_data[addr] = X / (X + Y);
                    break;
                }
                else {
                    double logX = log(U) / a;
                    double logY = log(V) / b;
                    double logM = logX > logY ? logX : logY;
                    logX -= logM;
                    logY -= logM;

                    output_data[addr] = exp(logX - log(exp(logX) + exp(logY)));
                    break;
                }
            }
        }
    }
    else {
        double Ga = sample_gamma_internal(&global_states[idx], a_data[addr]);
        double Gb = sample_gamma_internal(&global_states[idx], b_data[addr]);
        output_data[addr] = Ga/(Ga + Gb);
    }
  }
}

#ifdef __cplusplus
extern "C" {
#endif

int polygamma_cuda_wrapped(int n, int input_strideHeight, int input_strideWidth, int output_strideHeight, int output_strideWidth, int height, int width, float *input_data, float *output_data) {
  polygamma_cuda_kernel<<<1, NUM_BLOCKS>>>(n, input_strideHeight, input_strideWidth, output_strideHeight, output_strideWidth, height, width, input_data, output_data);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in polygamma_cuda_kernel: %s\n", hipGetErrorString(err));
    return 1;
  }
  return 0;
}

int lgamma_cuda_wrapped(int input_strideHeight, int input_strideWidth, int output_strideHeight, int output_strideWidth, int height, int width, float *input_data, float *output_data) {
  lgamma_cuda_kernel<<<1, NUM_BLOCKS>>>(input_strideHeight, input_strideWidth, output_strideHeight, output_strideWidth, height, width, input_data, output_data);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in lgamma_cuda_kernel: %s\n", hipGetErrorString(err));
    return 1;
  }
  return 0;
}

int lbeta_cuda_wrapped(int a_strideHeight, int a_strideWidth, int b_strideHeight, int b_strideWidth, int output_strideHeight, int output_strideWidth, int height, int width, float *a_data, float *b_data, float *output_data) {
  lbeta_cuda_kernel<<<1, NUM_BLOCKS>>>(a_strideHeight, a_strideWidth, b_strideHeight, b_strideWidth, output_strideHeight, output_strideWidth, height, width, a_data, b_data, output_data);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in lbeta_cuda_kernel: %s\n", hipGetErrorString(err));
    return 1;
  }
  return 0;
}

int polygamma_cuda_dbl_wrapped(int n, int input_strideHeight, int input_strideWidth, int output_strideHeight, int output_strideWidth, int height, int width, double *input_data, double *output_data) {
  polygamma_cuda_dbl_kernel<<<1, NUM_BLOCKS>>>(n, input_strideHeight, input_strideWidth, output_strideHeight, output_strideWidth, height, width, input_data, output_data);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in polygamma_cuda_dbl_kernel: %s\n", hipGetErrorString(err));
    return 1;
  }
  return 0;
}

int lgamma_cuda_dbl_wrapped(int input_strideHeight, int input_strideWidth, int output_strideHeight, int output_strideWidth, int height, int width, double *input_data, double *output_data) {
  lgamma_cuda_dbl_kernel<<<1, NUM_BLOCKS>>>(input_strideHeight, input_strideWidth, output_strideHeight, output_strideWidth, height, width, input_data, output_data);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in lgamma_cuda_dbl_kernel: %s\n", hipGetErrorString(err));
    return 1;
  }
  return 0;
}

int lbeta_cuda_dbl_wrapped(int a_strideHeight, int a_strideWidth, int b_strideHeight, int b_strideWidth, int output_strideHeight, int output_strideWidth, int height, int width, double *a_data, double *b_data, double *output_data) {
  lbeta_cuda_dbl_kernel<<<1, NUM_BLOCKS>>>(a_strideHeight, a_strideWidth, b_strideHeight, b_strideWidth, output_strideHeight, output_strideWidth, height, width, a_data, b_data, output_data);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in lbeta_cuda_dbl_kernel: %s\n", hipGetErrorString(err));
    return 2;
  }
  return 0;
}

int sample_gamma_dbl_wrapped(int height, int width, double *a_data, double *output_data) {
  sample_gamma_dbl_kernel<<<1, NUM_BLOCKS>>>(height, width, a_data, output_data);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in sample_gamma_dbl_wrapped: %s\n", hipGetErrorString(err));
    return 2;
  }
  return 0;
}

int sample_beta_dbl_wrapped(int height, int width, double *a_data, double *b_data, double *output_data) {
  sample_beta_dbl_kernel<<<1, NUM_BLOCKS>>>(height, width, a_data, b_data, output_data);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in sample_beta_dbl_wrapped: %s\n", hipGetErrorString(err));
    return 2;
  }
  return 0;
}

void init_rand(void) {
    init<<<1, NUM_BLOCKS>>>(time(NULL));
}

#ifdef __cplusplus
}
#endif
